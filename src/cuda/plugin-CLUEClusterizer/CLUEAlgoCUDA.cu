#include "hip/hip_runtime.h"
#include <math.h>
#include <limits>
#include <iostream>

// GPU Add
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// for timing
#include <chrono>
#include <ctime>
// user include

#include "CLUEAlgoCUDA.h"
#include "CLUEAlgoKernels.h"
#include "CUDACore/cudaCheck.h"

void CLUEAlgoCUDA::init_device(int nPoints) {
  d_hist = cms::cuda::make_device_unique<LayerTilesCUDA[]>(NLAYERS, stream_);
  d_seeds = cms::cuda::make_device_unique<cms::cuda::VecArray<int, maxNSeeds>>(stream_);
  d_followers = cms::cuda::make_device_unique<cms::cuda::VecArray<int, maxNFollowers>[]>(nPoints, stream_);

  hist_ = d_hist.get();
  seeds_ = d_seeds.get();
  followers_ = d_followers.get();
}

void CLUEAlgoCUDA::setup(PointsCloud const& host_pc) {
  // copy input variables
  cudaCheck(hipMemcpyAsync(
      d_points.x.get(), host_pc.x.data(), sizeof(float) * host_pc.x.size(), hipMemcpyHostToDevice, stream_));
  cudaCheck(hipMemcpyAsync(
      d_points.y.get(), host_pc.y.data(), sizeof(float) * host_pc.x.size(), hipMemcpyHostToDevice, stream_));
  cudaCheck(hipMemcpyAsync(
      d_points.layer.get(), host_pc.layer.data(), sizeof(int) * host_pc.x.size(), hipMemcpyHostToDevice, stream_));
  cudaCheck(hipMemcpyAsync(
      d_points.weight.get(), host_pc.weight.data(), sizeof(float) * host_pc.x.size(), hipMemcpyHostToDevice, stream_));
  // initialize result and internal variables
  // // result variables
  //   cudaCheck(hipMemsetAsync(d_points.rho.get(), 0x00, sizeof(float) * host_pc.x.size(), stream_));
  //   cudaCheck(hipMemsetAsync(d_points.delta.get(), 0x00, sizeof(float) * host_pc.x.size(), stream_));
  //   cudaCheck(hipMemsetAsync(d_points.nearestHigher.get(), 0x00, sizeof(int) * host_pc.x.size(), stream_));
  //   cudaCheck(hipMemsetAsync(d_points.clusterIndex.get(), 0x00, sizeof(int) * host_pc.x.size(), stream_));
  //   cudaCheck(hipMemsetAsync(d_points.isSeed.get(), 0x00, sizeof(int) * host_pc.x.size(), stream_));
  // algorithm internal variables
  //   cudaCheck(hipMemsetAsync(d_hist.get(), 0x00, sizeof(LayerTilesCUDA) * NLAYERS, stream_));
  cudaCheck(hipMemsetAsync(d_seeds.get(), 0x00, sizeof(cms::cuda::VecArray<int, maxNSeeds>), stream_));
  //   cudaCheck(hipMemsetAsync(
  //       d_followers.get(), 0x00, sizeof(cms::cuda::VecArray<int, maxNFollowers>) * host_pc.x.size(), stream_));

  const dim3 blockSize(1024, 1, 1);
  dim3 gridSize(ceil(host_pc.x.size() / static_cast<float>(blockSize.x)), 1, 1);
  kernel_reset_followers<<<gridSize, blockSize, 0, stream_>>>(followers_, host_pc.x.size());
  gridSize.x = std::ceil(LayerTilesConstants::nRows * LayerTilesConstants::nColumns / static_cast<float>(blockSize.x));
  kernel_reset_hist<<<gridSize, blockSize, 0, stream_>>>(hist_);
}

void CLUEAlgoCUDA::makeClusters(PointsCloud const& host_pc) {
  setup(host_pc);
  ////////////////////////////////////////////
  // calculate rho, delta and find seeds
  // 1 point per thread
  ////////////////////////////////////////////
  const dim3 blockSize(1024, 1, 1);
  const dim3 gridSize(ceil(host_pc.x.size() / static_cast<float>(blockSize.x)), 1, 1);
  kernel_compute_histogram<<<gridSize, blockSize, 0, stream_>>>(hist_, d_points.view(), host_pc.x.size());
  kernel_calculate_density<<<gridSize, blockSize, 0, stream_>>>(hist_, d_points.view(), dc_, host_pc.x.size());
  kernel_calculate_distanceToHigher<<<gridSize, blockSize, 0, stream_>>>(
      hist_, d_points.view(), outlierDeltaFactor_, dc_, host_pc.x.size());
  kernel_find_clusters<<<gridSize, blockSize, 0, stream_>>>(
      seeds_, followers_, d_points.view(), outlierDeltaFactor_, dc_, rhoc_, host_pc.x.size());

  ////////////////////////////////////////////
  // assign clusters
  // 1 point per seeds
  ////////////////////////////////////////////
  const dim3 gridSize_nseeds(ceil(maxNSeeds / static_cast<float>(blockSize.x)), 1, 1);
  kernel_assign_clusters<<<gridSize_nseeds, blockSize, 0, stream_>>>(seeds_, followers_, d_points.view());
  hipStreamSynchronize(stream_);
}
